#include "CubicDynArr.h"

template<>template<>
__host__ hipResourceDesc CubicDynArrTexReader<int>::make_rd<int>(int* ptr, size_t bsize) {
    hipResourceDesc resDesc;
    memset(&resDesc, 0, sizeof(resDesc));
    resDesc.resType = hipResourceTypeLinear;
    resDesc.res.linear.devPtr = ptr;
    resDesc.res.linear.desc.f = hipChannelFormatKindSigned;
    resDesc.res.linear.desc.x = 32; // bits per channel
    resDesc.res.linear.desc.y = 0;
    resDesc.res.linear.desc.z = 0;
    resDesc.res.linear.desc.w = 0;
    resDesc.res.linear.sizeInBytes = bsize;
    return resDesc;
}

template<>template<>
__host__ hipResourceDesc CubicDynArrTexReader<unsigned int>::make_rd<unsigned int>(unsigned int* ptr, size_t bsize) {
    hipResourceDesc resDesc;
    memset(&resDesc, 0, sizeof(resDesc));
    resDesc.resType = hipResourceTypeLinear;
    resDesc.res.linear.devPtr = ptr;
    resDesc.res.linear.desc.f = hipChannelFormatKindUnsigned;
    resDesc.res.linear.desc.x = 32; // bits per channel
    resDesc.res.linear.desc.y = 0;
    resDesc.res.linear.desc.z = 0;
    resDesc.res.linear.desc.w = 0;
    resDesc.res.linear.sizeInBytes = bsize;
    return resDesc;
}