#include "hip/hip_runtime.h"
#include "CellManager.h"
#include ""
#include "cuda_helper_misc.h"
__device__ inline real ca2p_by_ext_stim(real _ext_stim) {
    

    return kbc * _ext_stim*_ext_stim * Cout / (Hb + _ext_stim*_ext_stim);
}

__device__ inline real ca2p_into_storage(real ca2p_in_cell) {
    
    return v_gamma * ca2p_in_cell / (kg + ca2p_in_cell);
}

__device__ inline real ER_domain1_active(real ip3) {

    return (mu0 + mu1 * ip3 / (ip3 + kmu));
}

__device__ inline real ER_domain2_active(real ca2p) {

    return (para_b + para_bb * ca2p / (para_k1 + ca2p));
}

__device__ inline real ca2p_reaction_factor(real u, real ER_domain3_deactive, real p, real B)
{
    

    const real ca2p_from_storage = k_flux * ER_domain1_active(p) * ER_domain2_active(u) * ER_domain3_deactive;

    return
        ca2p_from_storage
        - ca2p_into_storage(u)
        + leak_from_storage
        + ca2p_by_ext_stim(B);

}

__device__ inline real calc_th(bool is_alive, real agek) {

    
    //using namespace cont;


    return is_alive ?
        thgra + ((thpri - thgra)*real(0.5)) * (real(1.0) + tanh((THRESH_SP - agek) / delta_th)) :
        thpri;
}

__device__ inline real calc_Kpa(bool is_alive, real agek) {

    
    return is_alive ?
        Kgra + ((Kpri - Kgra)*real(0.5)) * (real(1.0) + tanh((THRESH_SP - agek) / delta_K)) :
        Kpri;
}

__device__ inline real calc_IAG(bool is_alive, real agek) {
    
    return is_alive ?
        real(0.5)*(real(1.0) + tanh((agek - THRESH_SP) / delta_I)) :
        iage_kitei;
}
__device__ inline real ex_inert_diff(real ca2p, real current_ex_inert, real _th) {
    
    return ((para_k2*para_k2 / (para_k2*para_k2 + ca2p*ca2p) - current_ex_inert) / _th);
}

__device__ inline real IP3_default_diff(real _Kpa, real a_avg, real current_IP3) {
    

    return (_Kpa*a_avg / (H0 + a_avg) - Kpp*current_IP3);
}

__device__ inline real fw(real diff, real w)
{

    return 0.5 - w + 0.5*tanh((wd - diff) / epsw0); //epsw0 == 0.1 //done
                                                            //-1. <-????
}

__global__ void dead_IP3_calc(const real* IP3_in,NonMembConn*nmconn,CellIterateRange_device cir,real* IP3_out) {
    const int index = threadIdx.x + blockIdx.x*blockDim.x;
    if (index >= cir.size<CI_DEAD>()) return;
    const int raw_idx = cir.idx<CI_DEAD>(index);
    const int al_head = cir.nums[CS_alive_hd];
    const int al_end = cir.nums[CS_musume_hd];
    NonMembConn& nmc = nmconn[raw_idx];
    const size_t sz = nmc.conn.size();
    real tmp = 0.0;
    const real myip3 = IP3_in[raw_idx];
    for (int i = 0; i < sz; i++) {
        const int opidx = nmc.conn[i];
        if (al_head <= opidx&&opidx < al_end) {
            tmp += IP3_in[opidx] - myip3;
        }
    }
    IP3_out[raw_idx]=myip3+ DT_Ca*(dp*tmp - Kpp*myip3);
}

struct gj_data {
    real data[CELL_CONN_NUM];
    __device__ real& operator[](int idx) {
        return data[idx];
    }
    __device__ const real& operator[](int idx)const {
        return data[idx];
    }
};

__global__ void supra_calc(const CellPos*cpos, 
    const real* ca2p_in,real* ca2p_out,
    real* ca2p_avg_out,
    real*diff_u_out, gj_data* gj, 
    real* exinert,
    const real* IP3_in,real* IP3_out,
    NonMembConn*nmconn, const real* agekset,const hipSurfaceObject_t ATP_first, const hipSurfaceObject_t ext_stim_first, CellIterateRange_device cir) {

	//(cpos,ca2p1,ca2p2,ca2p_avg,diffu,gj,exinert,IP3_1,IP3_2,nmc,agek,ATP_1,extstim,cir)
	const int index = threadIdx.x + blockIdx.x*blockDim.x;
    if (index >= cir.size<CI_ALIVE,CI_MUSUME,CI_FIX>()) return;
    const int raw_idx = cir.idx<CI_ALIVE, CI_MUSUME, CI_FIX>(index);
    const real4 mypos = cpos[raw_idx];
    const int ix = int(mypos.x*inv_dx);
    const int iy = int(mypos.y*inv_dy);
    const int iz = int(mypos.z*inv_dz);
    const real myagek = agekset[raw_idx];
    const bool is_alive = cir.nums[CS_alive_hd] <= raw_idx&&raw_idx < cir.nums[CS_musume_hd];
    const real _th = calc_th(is_alive,myagek );
    const real _Kpa = calc_Kpa(is_alive, myagek);
    const real IAGv = calc_IAG(is_alive, myagek);
    NonMembConn& nmc = nmconn[raw_idx];
    const size_t sz = nmc.conn.size();
    real tmp_diffu = 0.0;
    real tmp_IP3 = 0.0;
    //real tmp_gj = 0.0;
    const real myca2p = ca2p_in[raw_idx];
    gj_data& mygj = gj[raw_idx];

    for (int i = 0; i < sz; i++) {
        const int opidx = nmc.conn[i];
        //st==ALIVE||st==DEAD||st==FIX||st==MUSUME
        //MEMB-FIX-DER-AIR-DEAD-ALIVE-MUSUME
        const bool criteria_all = cir.nums[CS_fix_hd] <= opidx && !(cir.nums[CS_der_hd] <= opidx&&opidx < cir.nums[CS_dead_hd]);
        const bool criteria_alive = cir.nums[CS_alive_hd] <= opidx && opidx<cir.nums[CS_musume_hd];
        if (criteria_all) {
            tmp_diffu += mygj[i] * (ca2p_in[opidx] - myca2p);
            tmp_IP3 += mygj[i] * (IP3_in[opidx] - IP3_in[raw_idx]);
            if (criteria_alive) {
            	//if(mygj[i]==0.0)printf("PogChanhCBhh %f\n",mygj[i]);
                mygj[i] +=DT_Ca* fw(fabs(ca2p_in[opidx] - myca2p), mygj[i]);
            }
        }
    }
    const real extf=grid_avg8_sobj(ext_stim_first, ix, iy, iz);
   const real duo= diff_u_out[raw_idx]= ca2p_reaction_factor(myca2p, exinert[raw_idx], IP3_in[raw_idx],
       extf) + ca2p_du*IAGv*tmp_diffu;
   //if(index==0)printf("duo:%f tmpdiff:%f extf:%f test:%f %d %d %d\n",duo,tmp_diffu,extf,surf3Dread_real(ext_stim_first,ix,iy,iz),ix,iy,iz);
   IP3_out[raw_idx] =
       IP3_in[raw_idx] + DT_Ca*(IP3_default_diff(_Kpa, grid_avg8_sobj(ATP_first, ix, iy, iz), IP3_in[raw_idx]) + dp*IAGv*tmp_IP3);
   //if(index==0)printf("ip3O:%f\n",IP3_out[raw_idx]);
   const real cs = myca2p + DT_Ca*duo;
   ca2p_out[raw_idx] = cs;
   ca2p_avg_out[raw_idx] += cs;
   exinert[raw_idx]+=DT_Ca*ex_inert_diff(myca2p, exinert[raw_idx], _th);

   // c->IP3.set_next(c->IP3() + DT_Ca*(IP3_default_diff(_Kpa, grid_avg8(ATP_first(), ix, iy, iz), c->IP3()) + dp*IAGv*tmp_IP3));

}
__device__ inline real fa(real diffu, real A) {
    
    //using namespace cont;
    return STIM11*min0(diffu) - A*Kaa;
}
#define CDIM (4)
__global__ void ATP_refresh(const hipTextureObject_t cmap1,
    const hipTextureObject_t cmap2,
    const NonMembConn*nmc,CellIterateRange_device cir,
    const real* diffu_map,
    const hipSurfaceObject_t ATP_in, hipSurfaceObject_t ATP_out,
    const real*zzmax) {
    const int iz_bound = (int)((*zzmax + FAC_MAP*NON_MEMB_RAD) *inv_dz);

    const int z = threadIdx.z + blockIdx.z*CDIM;
    if (z >= iz_bound)return;
    const int x = threadIdx.x + blockIdx.x*CDIM;
    if (x >= NX)return;
    const int y = threadIdx.y + blockIdx.y*CDIM;
    if (y >= NY)return;

    const int prev_x = x == 0 ? NX - 1 : x - 1; const int next_x = x == NX - 1 ? 0 : x + 1;
    const int prev_y = y == 0 ? NY - 1 : y - 1; const int next_y = y == NY - 1 ? 0 : y + 1;
    const int prev_z = z == 0 ? 1 : z - 1; const int next_z = z == NZ - 1 ? NZ - 2 : z + 1;
#define midx(xx,yy,zz) (xx+NX*(yy+NY*zz))
    const int cidx = tex1Dfetch<int>(cmap1, midx(x, y, z));
    //MEMB-FIX-DER-AIR-DEAD-ALIVE-MUSUME
    const bool al_fix_mu = cir.nums[CS_fix_hd]<=cidx&&!(cir.nums[CS_der_hd]<=cidx&&cidx<cir.nums[CS_alive_hd]);
    const real diffu = al_fix_mu ? diffu_map[cidx]:0.0;

    real asf=0.0;
    if(cidx>=0){
    	const NonMembConn& mynmc=nmc[cidx];
    	const size_t mysz=mynmc.conn.size();
    	for(int i=0;i<mysz;i++){
    		const int opidx=mynmc.conn[i];
    		if(cir.nums[CS_air_hd]<=opidx&&opidx<cir.nums[CS_dead_hd]){
    			asf=AIR_STIM;
    			break;
    		}
    	}
    }
    const real myv = surf3Dread_real(ATP_in, x, y, z);
    const real outv = myv +
        DT_Ca*(Da*(
            intmask_real(tex1Dfetch<int>(cmap2, midx(prev_x, y, z)),(surf3Dread_real(ATP_in, prev_x, y, z) - myv))
            + intmask_real(tex1Dfetch<int>(cmap2, midx(next_x, y, z)),(surf3Dread_real(ATP_in, next_x, y, z) - myv))
            + intmask_real(tex1Dfetch<int>(cmap2, midx(x, prev_y, z)),(surf3Dread_real(ATP_in, x, prev_y, z) - myv))
            + intmask_real(tex1Dfetch<int>(cmap2, midx(x, next_y, z)),(surf3Dread_real(ATP_in, x, next_y, z) - myv))
            + intmask_real(tex1Dfetch<int>(cmap2, midx(x, y, prev_z)),(surf3Dread_real(ATP_in, x, y, prev_z) - myv))
            + intmask_real(tex1Dfetch<int>(cmap2, midx(x, y, next_z)),(surf3Dread_real(ATP_in, x, y, next_z) - myv))
            )*inv_dx*inv_dx
            + fa(diffu, myv)+asf);
    //if(x==NX/2&&y==NY/2&&z==NZ/4)printf("myv:%f\n",myv);
    surf3Dwrite_real(outv, ATP_out, x, y, z);
}
__global__ void test22() {
    printf("LUL\n");
}
__global__ void initialize_ca2p_calc(CellIterateRange_device cir,CellAttr*cat,
		gj_data*gj,real* ca2p1,real* ca2p2,real* ca2p_avg,real* diffu,real*exinert,real*IP3_1,real*IP3_2,real*agek) {
const int index=threadIdx.x+blockIdx.x*blockDim.x;
if(index>=cir.nums[CS_asz])return;
ca2p1[index]=cat[index].ca2p_avg;
ca2p2[index]=cat[index].ca2p_avg;
ca2p_avg[index]=real(0.0);
diffu[index]=real(0.0);
exinert[index]=cat[index].ex_inert;
IP3_1[index]=cat[index].IP3;
IP3_2[index]=cat[index].IP3;
agek[index]=cat[index].agek;
for(int i=0;i<CELL_CONN_NUM;i++){
	gj[index][i]=0.0;//gj_init;
}
}
__global__ void initialize_ATP(hipSurfaceObject_t ATP_1,hipSurfaceObject_t ATP_2) {
surf3Dwrite_real(real(0.0),ATP_1,threadIdx.x,blockIdx.x,blockIdx.y);
surf3Dwrite_real(real(0.0),ATP_2,threadIdx.x,blockIdx.x,blockIdx.y);
}
template<typename T>
__device__ void swap_POD(T& s1,T& s2){
	T tmp=s1;
	s1=s2;
	s2=tmp;
}
__global__ void finalize_ca2p(CellIterateRange_device cir,CellAttr*cat,real* ca2p_avg,real*exinert,real*IP3){
	const int index=threadIdx.x+blockIdx.x*blockDim.x;
	if(index>=cir.nums[CS_asz])return;
	cat[index].ca2p_avg=ca2p_avg[index]/Ca_ITR;
	//if(ca2p_avg[index]!=0.0&&ca2p_avg[index]<200.0)printf("much:%d %f %f %f %f\n",index,cat[index].ca2p_avg,ca2p_avg[index],IP3[index],exinert[index]);
	cat[index].ex_inert=exinert[index];
	cat[index].IP3=IP3[index];
}
__global__ void ca2p_proc_parent(
		CellAttr*cat,NonMembConn*nmc,CellPos*cpos,hipSurfaceObject_t extstim,hipTextureObject_t cmap1,hipTextureObject_t cmap2,
		const real*zzmax,
		hipSurfaceObject_t ATP_1,hipSurfaceObject_t ATP_2,
		CellIterateRange_device cir,gj_data*gj,
		real* ca2p1,real* ca2p2,real* ca2p_avg,real* diffu,real*exinert,real*IP3_1,real*IP3_2,real*agek){
	if(!(threadIdx.x==0&&blockIdx.x==0))return;
	if(cir.nums[CS_count_sw]>=SW_THRESH||cir.nums[CS_count_num_sc]>0){
		printf("ca2p calc...\n");
		initialize_ca2p_calc<<<cir.nums[CS_asz]/256+1,256>>>
				(cir,cat,gj,ca2p1,ca2p2,ca2p_avg,diffu,exinert,IP3_1,IP3_2,agek);
		initialize_ATP<<<dim3(NY,NZ),NX>>>(ATP_1,ATP_2);
		hipDeviceSynchronize();
		for(unsigned int i=0;i<Ca_ITR;i++){
		dead_IP3_calc<<<cir.size<CI_DEAD>()/64+1,64>>>(IP3_1,nmc,cir,IP3_2);

		supra_calc<<<cir.size<CI_ALIVE,CI_MUSUME,CI_FIX>()/64+1,64>>>(cpos,ca2p1,ca2p2,ca2p_avg,diffu,gj,exinert,IP3_1,IP3_2,nmc,agek,ATP_1,extstim,cir);
		ATP_refresh<<<dim3(NX/CDIM+1,NY/CDIM+1,NZ/CDIM+1),dim3(CDIM,CDIM,CDIM)>>>(cmap1,cmap2,nmc,cir,diffu,ATP_1,ATP_2,zzmax);
		hipDeviceSynchronize();
		_wrap_bound<<<dim3(NY, NZ), NX >>>(ATP_2);
		hipDeviceSynchronize();
		swap_POD(ATP_1,ATP_2);
		swap_POD(ca2p1,ca2p2);
		swap_POD(IP3_1,IP3_2);
		hipDeviceSynchronize();
		const int tidx=cir.nums[CS_musume_hd];
		if(i==Ca_ITR-1){

			//printf("check ca2p_avg %f %f %f %f\n",ca2p_avg[tidx],IP3_1[tidx],exinert[tidx],gj[tidx-1][0]);
		}

		}
		const int tidx=cir.nums[CS_musume_hd];
		//printf("checklast %d ca2p_avg %f %f %f %f\n",tidx,ca2p_avg[tidx],IP3_1[tidx],exinert[tidx],gj[tidx-1][0]);
		finalize_ca2p<<<cir.nums[CS_asz]/256+1,256>>>
						(cir,cat,ca2p_avg,exinert,IP3_1);
		cir.nums[CS_count_sw]=0;
		if(cir.nums[CS_count_num_sc]>0)cir.nums[CS_count_num_sc]--;
	}
}

void calc_ca2p(CellManager&cm,const hipSurfaceObject_t extstim,hipTextureObject_t cmap1,hipTextureObject_t cmap2) {
	const size_t ubs=2*cm.all_size();
    static thrust::device_vector<real> ca2p1(ubs), ca2p2(ubs),
ca2p_avg(ubs), diffu(ubs), exinert(ubs), IP3_1(ubs), IP3_2(ubs),agek(ubs);
    static thrust::device_vector<gj_data> gj(ubs);
    static cuda3DSurface<real> ATP_1(NX, NY, NZ), ATP_2(NX, NY, NZ);
    static std::vector<thrust::device_vector<real>*> vptr={&ca2p1,&ca2p2,&ca2p_avg,&diffu,&exinert,&IP3_1,&IP3_2,&agek};
    //retarded
    for(auto& vp:vptr){
    	while((double)((int)vp->size()-(int)cm.all_size())<(double)vp->size()*0.2){
    		vp->resize(vp->size()*1.5);
    	}
    }

    while((double)((int)gj.size()-(int)cm.all_size())<(double)gj.size()*0.2){
    	gj.resize(gj.size()*1.5);
        	}
#define _dptr(v) thrust::raw_pointer_cast(v.data())

    ca2p_proc_parent<<<1,32>>>(cm.get_device_attr(),cm.get_device_all_nm_conn(),cm.get_device_pos_all(),extstim,cmap1,cmap2,cm.zzmax_ptr(),
    		ATP_1.st,ATP_2.st,cm.get_cell_iterate_range_d(),
    		_dptr(gj),
    		_dptr(ca2p1),_dptr(ca2p2),_dptr(ca2p_avg),_dptr(diffu),_dptr(exinert),_dptr(IP3_1),_dptr(IP3_2),_dptr(agek));
    //hipDeviceSynchronize();
}
