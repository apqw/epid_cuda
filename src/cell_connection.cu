#include "hip/hip_runtime.h"
#include "define.h"
#include "CubicDynArr.h"
#include "LFStack.h"
#include ""
#include "cuda_helper_misc.h"
#include "math_helper.h"
#include "cell_connection.h"
#include "CellManager.h"
#include <cfloat>
#include <thrust/count.h>

/** �O���b�h�T�C�Y*/
static constexpr real AREA_GRID_ORIGINAL = 2.0;//ok

                                                 /** �O���b�h�T�C�Y (?)*/
static constexpr real AREA_GRID = AREA_GRID_ORIGINAL + real(1e-7);//ok
static constexpr real AREA_GRID_INV = real(1.0)/AREA_GRID;//ok

                                                              /** X�����̃O���b�h��*/
static constexpr int	ANX = (int)((real)LX / AREA_GRID_ORIGINAL + 0.5);//ok
                                                                                  /** Y�����̃O���b�h��*/
static constexpr int	ANY = (int)((real)LY / AREA_GRID_ORIGINAL + 0.5);//ok
                                                                                  /** Z�����̃O���b�h��*/
static constexpr int	ANZ = (int)((real)LZ / AREA_GRID_ORIGINAL);//ok
                                                                            /** �O���b�h1������̍זE�i�[����� */
static constexpr int	N3 = 200; //max grid cell num //ok
                                  /** 1�זE�̐ڑ��ő吔 */
//static constexpr int	N2 = 400; //max conn num //ok

__global__ void grid_init(hipTextureObject_t pos_tex,CubicDynArrAccessor<LFStack<int, N3>> darr,size_t sz) {
    //need memset
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < sz) {
        CellPos cp = tex1Dfetch_real4(pos_tex, index);
        //printf("%f %f %f\n", cp.x, cp.y, cp.z);
        
        int aix=  (int)((real(0.5)*LX - p_diff_x(real(0.5)*LX, cp.x)) * AREA_GRID_INV);
        int aiy = (int)((real(0.5)*LY - p_diff_y(real(0.5)*LY, cp.y)) * AREA_GRID_INV);
        int aiz = (int)((min0(cp.z)) * AREA_GRID_INV);
        
        assert(!(aix >= ANX || aiy >= ANY || aiz >= ANZ || aix < 0 || aiy < 0 || aiz < 0));
        
        darr.at(aix, aiy, aiz).push_back_d(index);
        
    }
}

#define _m_max(a,b) ((a)>(b)?(a):(b))
#define _m_min(a,b) ((a)<(b)?(a):(b))

#define grid_init_THREAD_NUM (64)
#define connect_proc_THREAD_NUM (128)

#define TH_MULTI (32)
__global__ void connect_proc(hipTextureObject_t pos_tex,NonMembConn* all_nm_conn, CubicDynArrAccessor<LFStack<int, N3>> darr, int nmemb_start,size_t sz) {
    //need memset
    const int th_id = threadIdx.x%TH_MULTI;
    const int index = nmemb_start+blockIdx.x * (blockDim.x/TH_MULTI) + threadIdx.x/TH_MULTI;
    __shared__ real4 mycp[connect_proc_THREAD_NUM/TH_MULTI];
    static constexpr int srange = 2;
    static constexpr int width = srange*2+1;
    if (index < sz) {
        
        if (th_id > 24)return;
        /*
        if (th_id==0&&index%100==0)printf("PogAndChamp %d\n",index);
        if (th_id == 0 && index==sz-1)printf("End %d\n", index);
        */
        /*
        const CellPos cp = tex1Dfetch_real4(pos_tex, index);
        const int anx = (int)(cp.x * AREA_GRID_INV);
        const int any = (int)(cp.y* AREA_GRID_INV);
        const int anz = (int)(cp.z* AREA_GRID_INV);
        
        assert(!(anx >= (int)ANX || any >= (int)ANY || anz >= (int)ANZ || anx < 0 || any < 0 || anz < 0));
        //const int cw = width / TH_MULTI + th_id;
        const int xstart = anx -srange+th_id%width;
        const int xend = xstart;
        const int ystart = any - srange+th_id/width;
        const int yend = ystart;
        const int zstart = _m_max(anz - srange,0); //zstart = zstart > 0 ? zstart : 0;
        const int zend = _m_min(anz + srange, ANZ - 1); //zend = zend < ANZ - 1 ? zend : ANZ - 1;
        for (int j = xstart; j <= xend; j++) {
            const int cj = (j + ANX) % ANX;
            for (int k = ystart; k <= yend; k++) {
                const int ck = (k + ANY) % ANY;
                for (int cl = zstart; cl <= zend; cl++) {
                    //const int cl = l;
                    const LFStack<int, N3>& stref = darr.at(cj, ck, cl);
                    const size_t g_sz = stref.size();
                    for (int m = 0; m < g_sz; m++) {
                        const int oi = stref[m];
                        if (index <= oi)continue;
                        const CellPos ocp = tex1Dfetch_real4(pos_tex, oi);
                        const real rad_sum = oi >= nmemb_start ? NON_MEMB_RAD + NON_MEMB_RAD : MEMB_RAD + NON_MEMB_RAD;
                        if (p_dist_sq(cp, ocp) <= LJ_THRESH*LJ_THRESH*rad_sum*rad_sum) {
                            all_nm_conn[index].conn.push_back_d(oi);
                            all_nm_conn[oi].conn.push_back_d(index);
                        }
                    }
                }
            }
        }
        */
        if(th_id==0)mycp[threadIdx.x / TH_MULTI] = tex1Dfetch_real4(pos_tex, index);
        __syncthreads();
        const CellPos& cp = mycp[threadIdx.x / TH_MULTI];
        const int anx = (int)(cp.x * AREA_GRID_INV);
        const int any = (int)(cp.y* AREA_GRID_INV);
        const int anz = (int)(cp.z* AREA_GRID_INV);

        assert(!(anx >= (int)ANX || any >= (int)ANY || anz >= (int)ANZ || anx < 0 || any < 0 || anz < 0));
        //const int cw = width / TH_MULTI + th_id;
        const int j = anx - srange + th_id%width;
        const int k = any - srange + th_id / width;
        const int zstart = _m_max(anz - srange, 0); //zstart = zstart > 0 ? zstart : 0;
        const int zend = _m_min(anz + srange, ANZ - 1); //zend = zend < ANZ - 1 ? zend : ANZ - 1;
            const int cj = (j + ANX) % ANX;
                const int ck = (k + ANY) % ANY;
                for (int cl = zstart; cl <= zend; cl++) {
                    //const int cl = l;
                    const LFStack<int, N3>& stref = darr.at(cj, ck, cl);
                    const size_t g_sz = stref.size();
                    for (int m = 0; m < g_sz; m++) {
                        const int oi = stref[m];
                        if (index <= oi)continue;
                        const CellPos ocp = tex1Dfetch_real4(pos_tex, oi);
                        const real rad_sum = oi >= nmemb_start ? NON_MEMB_RAD + NON_MEMB_RAD : MEMB_RAD + NON_MEMB_RAD;
                        if (p_dist_sq(cp, ocp) <= LJ_THRESH*LJ_THRESH*rad_sum*rad_sum) {
                            all_nm_conn[index].conn.push_back_d(oi);
                            all_nm_conn[oi].conn.push_back_d(index);
                        }
                    }
                }
            
        
    }
}
/*
__global__ void connect_proc2(hipTextureObject_t pos_tex, NonMembConn* all_nm_conn, int nmemb_start, size_t sz) {
    const int index = nmemb_start + blockIdx.x * blockDim.x + threadIdx.x;
    static constexpr int srange = 2;
    if (index < sz) {
        const NonMembConn& nmc = all_nm_conn[index];
        const size_t csz = nmc.conn.size();
        for (int i = 0; i < csz; i++) {
            const int oi = nmc.conn[i];
            if (oi < nmemb_start) {
                all_nm_conn[oi].conn.push_back_d(index);
            }
        }
    }
}
*/
__global__ void find_dermis(hipTextureObject_t pos_tex, CellAttr* nm_cattr, const NonMembConn* all_nm_conn,const CellIndex* fix_musume_filtered,size_t nm_start,size_t sz) {
    const int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < sz) {
        //const int raw_index = index - nmemb_start;
        const int raw_idx = fix_musume_filtered[index];
            real d1sq = REAL_MAX;
            //real distsq = real{ 0.0 };
            int dermis_idx = -1;
            const NonMembConn& nmc = all_nm_conn[raw_idx];
            const int nmc_sz = nmc.conn.size();
            for (int i = 0; i < nmc_sz; i++) {
                const int raw_conn_idx = nmc.conn[i];
                if (raw_conn_idx  < nm_start) {//memb
                    const real distsq = p_dist_sq(tex1Dfetch_real4(pos_tex, raw_conn_idx), tex1Dfetch_real4(pos_tex, raw_idx));
                    if (distsq < d1sq) {
                        d1sq = distsq;
                        dermis_idx = raw_conn_idx;
                    }
                }

            }
            nm_cattr[raw_idx-nm_start].dermis = dermis_idx;
    }
}




void connect_cell(CellManager & cman)
{
    static CubicDynArrGenerator<LFStack<int, N3>> area(ANX, ANY, ANZ);
    area.memset_zero();
    cman.clear_all_non_memb_conn_both();

    size_t sz = cman.all_size();
    size_t nm_start = cman.memb_size();

    grid_init<<<((unsigned int)sz)/ grid_init_THREAD_NUM +1, grid_init_THREAD_NUM >>>(cman.get_pos_tex(), area.acc, sz);
    CUDA_SAFE_CALL(hipDeviceSynchronize());
    connect_proc << <TH_MULTI*unsigned(sz- nm_start) / connect_proc_THREAD_NUM + 1, connect_proc_THREAD_NUM >> >(cman.get_pos_tex(),cman.get_device_all_nm_conn(), area.acc, int(nm_start), sz);
    CUDA_SAFE_CALL(hipDeviceSynchronize());
   // connect_proc2 << <(sz - nm_start) / connect_proc_THREAD_NUM + 1, connect_proc_THREAD_NUM >> >(cman.get_pos_tex(), cman.get_device_all_nm_conn(), nm_start, sz);
   // CUDA_SAFE_CALL(hipDeviceSynchronize());

    int flt_num = 0;
    const CellIndex* ci=cman.nm_filter.filter_by_state<FIX, MUSUME>(&flt_num);
    find_dermis << <flt_num / 64 + 1, 64 >> > (cman.get_pos_tex(), cman.get_device_nmattr(), cman.get_device_all_nm_conn(), ci, nm_start, flt_num);
    CUDA_SAFE_CALL(hipDeviceSynchronize());

}