#include "hip/hip_runtime.h"
#include "cuda_helper_misc.h"

template<> template<>
void cuda3DSurface<double>::init<double>(size_t w, size_t h, size_t d) {
    hipChannelFormatDesc cdc = hipCreateChannelDesc<int2>();

    hipMalloc3DArray(&arr, &cdc, make_hipExtent(w, h, d));
    hipResourceDesc resDesc;
    memset(&resDesc, 0, sizeof(resDesc));
    resDesc.resType = hipResourceTypeArray;
    resDesc.res.array.array = arr;


    hipCreateSurfaceObject(&st, &resDesc);
}

__global__ void _wrap_bound(hipTextureObject_t data) {
    if (threadIdx.x >= NX)return;
    const real r1 = surf3Dread_real(data, threadIdx.x, 0, blockIdx.y);
    surf3Dwrite_real(r1, data, threadIdx.x, NY - 1, blockIdx.y);

    const real r2 = surf3Dread_real(data, NX - 1, blockIdx.x, blockIdx.y);
    surf3Dwrite_real(r2, data, 0, blockIdx.x, blockIdx.y);


}

void wrap_bound(hipTextureObject_t data) {
    _wrap_bound << <dim3(NY, NZ), NX >> > (data);
}

#ifdef USE_DOUBLE_AS_REAL
hipResourceDesc make_real4_resource_desc(CellPos*r4ptr, size_t len) {
    hipResourceDesc resDesc;
    memset(&resDesc, 0, sizeof(resDesc));
    resDesc.resType = hipResourceTypeLinear;
    resDesc.res.linear.devPtr = r4ptr;
    resDesc.res.linear.desc.f = hipChannelFormatKindSigned;
    resDesc.res.linear.desc.x = 32; // bits per channel
    resDesc.res.linear.desc.y = 32;
    resDesc.res.linear.desc.z = 32;
    resDesc.res.linear.desc.w = 32;
    resDesc.res.linear.sizeInBytes = len * sizeof(real4);
    return resDesc;
}


#else
hipResourceDesc make_real4_resource_desc(CellPos*r4ptr, size_t len) {
    hipResourceDesc resDesc;
    memset(&resDesc, 0, sizeof(resDesc));
    resDesc.resType = hipResourceTypeLinear;
    resDesc.res.linear.devPtr = r4ptr;
    resDesc.res.linear.desc.f = hipChannelFormatKindFloat;
    resDesc.res.linear.desc.x = 32; // bits per channel
    resDesc.res.linear.desc.y = 32;
    resDesc.res.linear.desc.z = 32;
    resDesc.res.linear.desc.w = 32;
    resDesc.res.linear.sizeInBytes = len * sizeof(real4);
    return resDesc;
}
#endif