#include "hip/hip_runtime.h"
#include "define.h"
#include "cell_movement.h"
#include "math_helper.h"
#include "CellManager.h"

#include ""
#include "cuda_helper_misc.h"


//////////////////// MEMB_BEND
__device__ inline real4 tri_normal(const real4 p1, const real4 p2, const real4 p3) {
    return normalize3_r_dev_w_inv(cross3(vsub(p1, p3), vsub(p2, p3)));
}

__device__ inline real4 tri_rho(const real4 n1,const real4 b1) {
    return vsub(b1, cvmul(dot3(n1, b1), n1));
}

__device__ inline real _memb_bend_diff_factor(const real cdot) {
    if (cdot >= real(0.0)) {
        return (real(1.0) - cdot);
    }
    else {
        return real(1.0) / ((real(1.0) + cdot)*(real(1.0) + cdot));
    }

}

__device__ real4 tri_bend_1(const real4 r1, const real4 a1, const real4 b1, const real4 c1) {
    //vec3 r1v = ctov(r1),a1v=ctov(a1),b1v=ctov(b1),c1v=ctov(c1);
    real4 norm1_pre = cross3(vpsub(r1, b1), vpsub(a1, b1));
    real norm1_pre_norm_inv;
    real4 norm1 = normalize3_r_dev_w_inv(norm1_pre, &norm1_pre_norm_inv), norm2 = normalize3_r_dev_w_inv(cross3(vpsub(a1, b1), vpsub(c1, b1)));
    real dot1 = dot3(norm1, norm2);
    return cvmul(_memb_bend_diff_factor(dot1), cvmul(norm1_pre_norm_inv,cross3(vpsub(a1, b1), tri_rho(norm1, norm2))));
}

__device__ real4 tri_bend_2(const real4 r1, const real4 a1, const real4 b1, const real4 c1) {
    //vec3 r1v = ctov(r1),a1v=ctov(a1),b1v=ctov(b1),c1v=ctov(c1);
    real4 norm1_pre = cross3(vpsub(r1, c1), vpsub(b1, c1));
    real4 norm2_pre = cross3(vpsub(r1, b1), vpsub(a1, b1));

    real norm1_pre_norm_inv, norm2_pre_norm_inv;
    real4 norm1 = normalize3_r_dev_w_inv(norm1_pre, &norm1_pre_norm_inv), norm2 = normalize3_r_dev_w_inv(norm2_pre, &norm2_pre_norm_inv);


    real dot1 = dot3(norm1, norm2);

    return cvmul(_memb_bend_diff_factor(dot1), (vadd(cvmul(norm1_pre_norm_inv,cross3(vpsub(b1, c1), tri_rho(norm1, norm2))), cvmul(norm2_pre_norm_inv,cross3(vpsub(a1, b1), tri_rho(norm2, norm1))))));
}

__global__ void  calc_memb_bend(hipTextureObject_t pos_tex,const MembConn* mconn,size_t sz,CellPos* out) {
    volatile int index = threadIdx.x + blockIdx.x*blockDim.x;
    if (index<sz) {
        //Memb* memb = mbptr + index;
        real4 dr = { real(0.0),real(0.0),real(0.0),real(0.0) };
        const CellPos mypos = tex1Dfetch_real4(pos_tex, index);
 
        for (int i = 0; i<6; i++) {
            //if(memb->md.memb_spr[i]>0.01 && memb->md.memb_spr[(i+1)%6]>0.01 )
            vadda(&dr, tri_bend_1(
                mypos,
                tex1Dfetch_real4(pos_tex, mconn[index].conn[i]),//mbptr + memb->adj_memb[i],
                tex1Dfetch_real4(pos_tex, mconn[index].conn[(i + 1) % 6]),
                tex1Dfetch_real4(pos_tex, mconn[mconn[index].conn[(i + 1) % 6]].conn[i])
            ));
                //mbptr + memb->adj_memb[(i + 1) % 6], mbptr + (mbptr + memb->adj_memb[(i + 1) % 6])->adj_memb[i]));

        }
        for (int i = 0; i<6; i++) {
            //if(memb->md.memb_spr[i]>0.01 && memb->md.memb_spr[(i+1)%6]>0.01 )
            vadda(&dr, tri_bend_2(
                mypos,
                tex1Dfetch_real4(pos_tex, mconn[index].conn[i]),
                tex1Dfetch_real4(pos_tex, mconn[index].conn[(i + 1) % 6]),
                tex1Dfetch_real4(pos_tex, mconn[index].conn[(i + 2) % 6])
            ));

        }
        out[index] = vadd(mypos, cvmul(DT_Cell*KBEND, dr));
    }
}

//////////////////// wall
template<CELL_STATE cst>
__device__ inline bool collide_with_wall(const real zpos) {
    return zpos < NON_MEMB_RAD;
}

template<>
__device__ inline bool collide_with_wall<MEMB>(const real zpos) {
    return zpos < MEMB_RAD;
}

__device__ inline real wall_interaction_memb(const real cz) {
    const real distlj = real(2.0)*cz;
    const real LJ6 = POW6(MEMB_RAD) / POW6(cz);
    //LJ6 = LJ6*LJ6;
    //LJ6 = LJ6*LJ6*LJ6;
    const real ljm = real(4.0)*eps_m*LJ6*(LJ6 - real(1.0)) / (distlj*distlj);
    return ljm*2.0*cz;
}

__device__ inline real wall_interaction(const real cz) {
    const real distlj = real(2.0)*cz;
    const real LJ6 = POW6(NON_MEMB_RAD) / POW6(cz);
    //LJ6 = LJ6*LJ6;
    //LJ6 = LJ6*LJ6*LJ6;
    const real ljm = real(4.0)*eps_m*LJ6*(LJ6 - real(1.0)) / (distlj*distlj);
    return ljm*2.0*cz;
}
struct c_memb_to_memb {
    __device__ real operator()(const real4 c1, const real4 c2)const {
        constexpr real cr_dist = MEMB_RAD_SUM*P_MEMB;
        constexpr real cr_dist_inv = 1.0 / cr_dist;
        constexpr real cr_dist_sq = cr_dist*cr_dist;


        const real dist_sq = p_dist_sq(c1, c2);
        if (dist_sq < cr_dist_sq) {
            const real LJ6 = POW3(cr_dist_sq) / POW3(dist_sq);
            return real(4.0)*eps_m*LJ6*(LJ6 - real(1.0)) / dist_sq;
        }
        else if (dist_sq < MEMB_RAD_SUM_SQ) {
            return -DER_DER_CONST*(cr_dist_inv - rsqrt(dist_sq));
            //-(DER_DER_CONST / distlj) * (distlj / cr_dist - 1.0);
        }
        else {
            const real distlj = sqrt(dist_sq);
            constexpr real lambda_dist = (real(1.0) + P_MEMB)*MEMB_RAD_SUM;
            const real LJ6 = POW6(cr_dist) / POW6(lambda_dist - distlj);
            return -(DER_DER_CONST / MEMB_RAD_SUM)*((real(1.0) - P_MEMB) / P_MEMB)
                - real(4.0) * eps_m*(LJ6*(LJ6 - real(1.0))) / ((lambda_dist - distlj)*distlj);
        }

    }
};

template<class Fn>
__device__ void cell_movement_calc(real4* accum_out, const real4& c1, const real4& c2, Fn calc_fn) {
    vadda(accum_out, cvmul(calc_fn(c1, c2), vpsub(c1, c2)));
}
__global__ void MEMB_interaction(hipTextureObject_t pos_tex,const MembConn* mconn,size_t sz, CellPos* out) {
    int index = threadIdx.x + blockIdx.x*blockDim.x;
    if (index >= sz)return;
    const CellPos mypos = tex1Dfetch_real4(pos_tex, index);
    real z_dis = real(0.0);
    real4 dr = { real(0.0), real(0.0), real(0.0), real(0.0) };
    if (collide_with_wall<MEMB>(mypos.z)) {
        z_dis += wall_interaction_memb(mypos.z);
    }

    for (int i = 0; i < MEMB_CONN_NUM; i++) {
        const real4 op = tex1Dfetch_real4(pos_tex, mconn[index].conn[i]);
        cell_movement_calc(&dr, mypos, op, c_memb_to_memb());
    }

    vadda(&out[index] , cvmul(DT_Cell, dr));

}

void calc_cell_movement(CellManager&cman) {
    size_t msz = cman.memb_size();
        calc_memb_bend << <msz / 32 + 1, 32 >> > (cman.get_pos_tex(), cman.get_device_mconn(), msz, cman.get_device_pos_all_out());
        hipDeviceSynchronize();
        MEMB_interaction << <msz / 64 + 1, 64 >> >(cman.get_pos_tex(), cman.get_device_mconn(), msz, cman.get_device_pos_all_out());
        cman.pos_swap_device();
        cman.refresh_pos_tex();
}